#include "hip/hip_runtime.h"
#include "SJPlenopticPacker.h"
#include "SJLog.h"
#include <hip/device_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <memory.h>
#include "opencv2/opencv.hpp"
using namespace std;
using namespace cv;
__global__ void convertDecodedToImageBuffer(const CUDA_UCHAR* src, CUDA_UCHAR* dst, SJDim imageW, SJDim imageH, SJDim imageW2, SJDim imageH2, int numCam, SJDim startIdx)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    int imgOffsetX = 0;
    int imgOffsetY = 0;
    int cam = startIdx;
    if (ix < imageW && iy < imageH) {

        while (true) {
            dst[(ix + iy * imageW + cam * imageW * imageH) * 3] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3];
            dst[(ix + iy * imageW + cam * imageW * imageH) * 3 + 1] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3 + 1];
            dst[(ix + iy * imageW + cam * imageW * imageH) * 3 + 2] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3 + 2];

            cam++;
            if (cam >= numCam) break;

            imgOffsetY += imageH;
            if (imgOffsetY >= imageH2) {
                imgOffsetX += imageW;
                imgOffsetY = 0;
            }
            if (imgOffsetX >= imageW2) {
                imgOffsetX = 0;
                break;
            }
        }
    }
}

__global__ void convertDecodedToImageBuffer_Resizing(const CUDA_UCHAR* src, CUDA_UCHAR* dst, SJDim imageW, SJDim imageH, SJDim newImageW, SJDim newImageH, SJDim imageW2, SJDim imageH2, int numCam, SJDim startIdx)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    int imgOffsetX = 0;
    int imgOffsetY = 0;
    int cam = startIdx;
    int factor = (int)(imageW / newImageW);
    if (ix < newImageW && iy < newImageH) {

        while (true) {
            dst[(ix + iy * newImageW + cam * newImageW * newImageH) * 3] = src[(ix * factor + imgOffsetX + (iy * factor + imgOffsetY) * imageW2) * 3 + 0];
            dst[(ix + iy * newImageW + cam * newImageW * newImageH) * 3 + 1] = src[(ix * factor + imgOffsetX + (iy * factor + imgOffsetY) * imageW2) * 3 + 1];
            dst[(ix + iy * newImageW + cam * newImageW * newImageH) * 3 + 2] = src[(ix * factor + imgOffsetX + (iy * factor + imgOffsetY) * imageW2) * 3 + 2];

            cam++;
            if (cam >= numCam) break;

            imgOffsetY += imageH;
            if (imgOffsetY >= imageH2) {
                imgOffsetX += imageW;
                imgOffsetY = 0;
            }
            if (imgOffsetX >= imageW2) {
                imgOffsetX = 0;
                break;
            }
        }
    }
}
__global__ void convertDecodedToAlphaBuffer(const CUDA_UCHAR* src, CUDA_UCHAR* alphaImage, int imageW, int imageH, int imageW2, int imageH2, int numLevel, int numCam, int startID, int levelID)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    int imgOffsetX = 0;
    int imgOffsetY = 0;
    int cam = startID;
    int level = levelID;
    if (ix < imageW && iy < imageH) {
        while (true) {
            if (level < numLevel - 2) {
                alphaImage[ix + iy * imageW + level * imageW * imageH + cam * imageW * imageH * numLevel] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3 + 0];
                alphaImage[ix + iy * imageW + (level + 1) * imageW * imageH + cam * imageW * imageH * numLevel] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3 + 1];
                alphaImage[ix + iy * imageW + (level + 2) * imageW * imageH + cam * imageW * imageH * numLevel] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3 + 2];
                level += 3;

                imgOffsetY += imageH;
                if (imgOffsetY >= imageH2) {
                    imgOffsetX += imageW;
                    imgOffsetY = 0;
                }
                if (imgOffsetX >= imageW2) {
                    imgOffsetX = 0;
                    break;
                }
            }
            else {
                alphaImage[ix + iy * imageW + level * imageW * imageH + cam * imageW * imageH * numLevel] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3 + 0];
                alphaImage[ix + iy * imageW + (level + 1) * imageW * imageH + cam * imageW * imageH * numLevel] = src[(ix + imgOffsetX + (iy + imgOffsetY) * imageW2) * 3 + 1];

                level = 0;
                cam++;
                if (cam >= numCam) break;

                imgOffsetY += imageH;
                if (imgOffsetY >= imageH2) {
                    imgOffsetX += imageW;
                    imgOffsetY = 0;
                }
                if (imgOffsetX >= imageW2) {
                    imgOffsetX = 0;
                    break;
                }
            }
        }
    }
}


SJPlenopticPacker::SJPlenopticPacker()
{
    m_pDim = NULL;
    m_pMPIDim = NULL;
    m_pLookupImage = NULL;
    m_pLookupAlphaImage = NULL;
    m_pLookupAlphaLevel = NULL;
}
SJPlenopticPacker::~SJPlenopticPacker()
{
    if (m_pDim) {
        delete[]m_pDim;
        m_pDim = NULL;
    }
    if (m_pMPIDim) {
        delete[]m_pMPIDim;
        m_pMPIDim = NULL;
    }
    if (m_pLookupImage) {
        delete[]m_pLookupImage;
        m_pLookupImage = NULL;
    }
    if (m_pLookupAlphaImage) {
        delete[]m_pLookupAlphaImage;
        m_pLookupAlphaImage = NULL;
    }
    if (m_pLookupAlphaLevel) {
        delete[]m_pLookupAlphaLevel;
        m_pLookupAlphaLevel = NULL;
    }
}
void SJPlenopticPacker::MakeLookupTableImage(int iDecWidth, int iDecHeight, SJDim numCam, SJDim width, SJDim height, int* pLookupImage)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    int imgOffsetX;
    int imgOffsetY;
    int camID;
    int levelID;
    int encoderID;
    encoderID = camID = imgOffsetX = imgOffsetY = 0;
    pLookupImage[encoderID] = camID;
    while (true) {

        camID++;
        if (camID >= numCam) break;

        imgOffsetY += height;
        if (imgOffsetY >= iDecHeight) {
            imgOffsetX += width;
            imgOffsetY = 0;
        }
        if (imgOffsetX >= iDecWidth) {
            imgOffsetX = 0;
            encoderID++;
            pLookupImage[encoderID] = camID;
        }
    }
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}
void SJPlenopticPacker::MakeLookupTableLayer(int iDecWidth, int iDecHeight, SJDim numCam, SJDim layerWidth, SJDim layerHeight, SJDim layerLevel, int* pLookupAlphaImage, int* pLookupAlphaLevel)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");
    int imgOffsetX;
    int imgOffsetY;
    int camID;
    int levelID;
    int encoderID;


    levelID = encoderID = camID = imgOffsetX = imgOffsetY = 0;
    int preCamID = -1;
    pLookupAlphaImage[encoderID] = camID;
    pLookupAlphaLevel[encoderID] = levelID;
    while (true) {
        if (levelID < layerLevel - 2) {

            imgOffsetY += layerHeight;

            levelID += 3;

            if (imgOffsetY >= iDecHeight) {
                imgOffsetX += layerWidth;
                imgOffsetY = 0;
            }
            if (imgOffsetX >= iDecWidth) {
                imgOffsetX = 0;
                encoderID++;

                pLookupAlphaImage[encoderID] = camID;
                pLookupAlphaLevel[encoderID] = levelID;
            }
        }

        else {
            imgOffsetY += layerHeight;

            levelID = 0;
            camID++;
            if (camID >= numCam) break;

            if (imgOffsetY >= iDecHeight) {
                imgOffsetX += layerWidth;
                imgOffsetY = 0;
            }
            if (imgOffsetX >= iDecWidth) {
                imgOffsetX = 0;
                encoderID++;

                pLookupAlphaImage[encoderID] = camID;
                pLookupAlphaLevel[encoderID] = levelID;
            }

        }
    }
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}


void SJPlenopticPacker::PackingImage(SJDim numCam, SJDim width, SJDim height, CPU_UCHAR** ppDecOut, CUDA_UCHAR** ppDecOutCUDA, int iDecWidth, int iDecHeight, int numImage, CPU_UCHAR* pImage)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    Mat matDstImg;
    Mat matSrcImg;
    //Mat tempImg;

    int imgOffsetX;
    int imgOffsetY;
    int camID;
    int levelID;
    int encoderID;

    encoderID = camID = imgOffsetX = imgOffsetY = 0;

    matDstImg = Mat(iDecHeight, iDecWidth, CV_8UC3, ppDecOut[encoderID]);
    memset(ppDecOut[encoderID], 0, iDecHeight * iDecWidth * 3);
    //tempImg = Mat::zeros(iDecHeight, iDecWidth, CV_8UC3);

    while (true) {

        matSrcImg = Mat(height, width, CV_8UC3, &pImage[camID * width * height * 3]);
        matSrcImg.copyTo(matDstImg(Rect(imgOffsetX, imgOffsetY, width, height)));
        camID++;
        if (camID >= numCam) {
            //cvtColor(tempImg, matDstImg, COLOR_RGB2BGR);
            break;
        }

        imgOffsetY += height;
        if (imgOffsetY >= iDecHeight) {
            imgOffsetX += width;
            imgOffsetY = 0;
        }
        if (imgOffsetX >= iDecWidth) {
            imgOffsetX = 0;
            //cvtColor(tempImg, matDstImg, COLOR_RGB2BGR);
            encoderID++;
            matDstImg = Mat(iDecHeight, iDecWidth, CV_8UC3, ppDecOut[encoderID]);
            memset(ppDecOut[encoderID], 0, iDecHeight * iDecWidth * 3);
            //tempImg = Mat::zeros(iDecHeight, iDecWidth, CV_8UC3);
        }
    }
    char filename[1024];
    for (int i = 0; i < numImage; i++) {
        hipMemcpy(ppDecOutCUDA[i], ppDecOut[i], iDecHeight * iDecWidth * 3 * sizeof(CUDA_UCHAR), hipMemcpyHostToDevice);
        //    matDstImg = Mat(iDecHeight, iDecWidth, CV_8UC3, ppDecOut[i]);
        //    sprintf_s(filename, "Image_%d.png", i);
        //    imwrite(filename, matDstImg);
    }
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}
void SJPlenopticPacker::PackingLayer(SJDim numCam, SJDim layerWidth, SJDim layerHeight, SJDim layerLevel, CPU_UCHAR** ppDecOut, CUDA_UCHAR** ppDecOutCUDA, int iDecWidth, int iDecHeight, int numLayer, CPU_UCHAR* pLayer)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    unsigned char* srcImage = new unsigned char[layerWidth * layerHeight * 3];

    Mat matDstImg;
    Mat matSrcImg;

    int index;
    int imgOffsetX;
    int imgOffsetY;
    int camID;
    int levelID;
    int encoderID;


    levelID = encoderID = camID = imgOffsetX = imgOffsetY = 0;
    int preCamID = -1;
    for (int i = 0; i < numLayer; i++) {
        memset(ppDecOut[i], 0, iDecWidth * iDecHeight * 3 * sizeof(unsigned char));
    }
    matDstImg = Mat(iDecHeight, iDecWidth, CV_8UC3, ppDecOut[encoderID]);
    while (true) {

        if (levelID < layerLevel - 2) {
            for (int j = 0; j < layerWidth * layerHeight; j++) {
                index = j + levelID * layerWidth * layerHeight + camID * layerWidth * layerHeight * layerLevel;
                srcImage[j * 3 + 0] = pLayer[index];
                srcImage[j * 3 + 1] = pLayer[index + layerWidth * layerHeight];
                srcImage[j * 3 + 2] = pLayer[index + layerWidth * layerHeight * 2];
            }
            matSrcImg = Mat(layerHeight, layerWidth, CV_8UC3, srcImage);

            matSrcImg.copyTo(matDstImg(Rect(imgOffsetX, imgOffsetY, layerWidth, layerHeight)));


            imgOffsetY += layerHeight;

            levelID += 3;

            if (imgOffsetY >= iDecHeight) {
                imgOffsetX += layerWidth;
                imgOffsetY = 0;
            }
            if (imgOffsetX >= iDecWidth) {
                imgOffsetX = 0;
                encoderID++;
                matDstImg = Mat(iDecHeight, iDecWidth, CV_8UC3, ppDecOut[encoderID]);
            }
        }

        else {
            for (int j = 0; j < layerWidth * layerHeight; j++) {
                index = j + levelID * layerWidth * layerHeight + camID * layerWidth * layerHeight * layerLevel;
                srcImage[j * 3 + 0] = pLayer[index];
                srcImage[j * 3 + 1] = pLayer[index + layerWidth * layerHeight];
                srcImage[j * 3 + 2] = 0;
            }
            matSrcImg = Mat(layerHeight, layerWidth, CV_8UC3, srcImage);

            matSrcImg.copyTo(matDstImg(Rect(imgOffsetX, imgOffsetY, layerWidth, layerHeight)));

            imgOffsetY += layerHeight;

            levelID = 0;
            camID++;
            if (camID >= numCam) break;

            if (imgOffsetY >= iDecHeight) {
                imgOffsetX += layerWidth;
                imgOffsetY = 0;
            }
            if (imgOffsetX >= iDecWidth) {
                imgOffsetX = 0;
                encoderID++;
                matDstImg = Mat(iDecHeight, iDecWidth, CV_8UC3, ppDecOut[encoderID]);

            }

        }
    }

    char filename[1024];
    for (int i = 0; i < numLayer; i++) {
        hipMemcpy(ppDecOutCUDA[i], ppDecOut[i], iDecHeight * iDecWidth * 3 * sizeof(CUDA_UCHAR), hipMemcpyHostToDevice);
        //    matDstImg = Mat(iDecHeight, iDecWidth, CV_8UC3, ppDecOut[i]);
        //    sprintf_s(filename, "Layer_%d.png", i);
        //    imwrite(filename, matDstImg);
    }
    delete[]srcImage;
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}

void SJPlenopticPacker::Initialize(SJDim* pDim, SJDim* pMPIDim, int iDecWidth, int iDecHeight, int numImage, int numLayer)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");
    m_iDecWidth = iDecWidth;
    m_iDecHeight = iDecHeight;
    m_numImage = numImage;
    m_numLayer = numLayer;

    m_pDim = new SJDim[4];
    m_pMPIDim = new SJDim[5];
    memcpy(m_pDim, pDim, 4 * sizeof(SJDim));
    memcpy(m_pMPIDim, pMPIDim, 5 * sizeof(SJDim));

    m_pLookupImage = new int[numImage];
    MakeLookupTableImage(iDecWidth, iDecHeight, pDim[0] * pDim[1], pDim[3], pDim[2], m_pLookupImage);

    m_pLookupAlphaImage = new int[numLayer];
    m_pLookupAlphaLevel = new int[numLayer];
    MakeLookupTableLayer(iDecWidth, iDecHeight, m_pMPIDim[0] * m_pMPIDim[1], m_pMPIDim[3], m_pMPIDim[2], m_pMPIDim[4], m_pLookupAlphaImage, m_pLookupAlphaLevel);

    m_layerThread.x = m_thread.x = BLOCKDIM_X;
    m_layerThread.y = m_thread.y = BLOCKDIM_Y;
    m_layerThread.z = m_thread.z = 1;

    m_grid.x = iDivUp(m_pDim[3], BLOCKDIM_X);
    m_layerGrid.x = iDivUp(m_pMPIDim[3], BLOCKDIM_X);
    m_grid.y = iDivUp(m_pDim[2], BLOCKDIM_Y);
    m_layerGrid.y = iDivUp(m_pMPIDim[2], BLOCKDIM_X);
    m_layerGrid.z = m_grid.z = 1;
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}


void SJPlenopticPacker::UnPackingImage(CUDA_UCHAR** ppDecOutCUDA, CUDA_UCHAR* pImageCUDA)
{
    //LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    for (int i = 0; i < m_numImage; i++) {
        convertDecodedToImageBuffer << <m_grid, m_thread >> > (ppDecOutCUDA[i], pImageCUDA, m_pDim[3], m_pDim[2], m_iDecWidth, m_iDecHeight, m_pDim[0] * m_pDim[1], m_pLookupImage[i]);
    }
    //LOGGING(LOG_LEVEL::VERBOSE, "End\n");
}

void SJPlenopticPacker::UnPackingImageWithIndex(CUDA_UCHAR* ppDecOutCUDA, CUDA_UCHAR* pImageCUDA, int index)
{
    //LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    convertDecodedToImageBuffer << <m_grid, m_thread >> > (ppDecOutCUDA, pImageCUDA, m_pDim[3], m_pDim[2], m_iDecWidth, m_iDecHeight, m_pDim[0] * m_pDim[1], m_pLookupImage[index]);
    //LOGGING(LOG_LEVEL::VERBOSE, "End\n");
}

void SJPlenopticPacker::UnPackingLayer(CUDA_UCHAR** ppDecOutCUDA, CUDA_UCHAR* pLayerCUDA)
{
    //dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    //dim3 grid(iDivUp(m_pMPIDim[3], BLOCKDIM_X), iDivUp(m_pMPIDim[2], BLOCKDIM_Y));

    for (int i = 0; i < m_numLayer; i++) {
        convertDecodedToAlphaBuffer << <m_layerGrid, m_layerThread >> > (ppDecOutCUDA[i], pLayerCUDA, m_pMPIDim[3], m_pMPIDim[2], m_iDecWidth, m_iDecHeight, m_pMPIDim[4], m_pMPIDim[0] * m_pMPIDim[1], m_pLookupAlphaImage[i], m_pLookupAlphaLevel[i]);
    }
}

void SJPlenopticPacker::UnPackingImageCPU(SJDim numCam, SJDim width, SJDim height, const CPU_UCHAR** ppDecOut, int iDecWidth, int iDecHeight, int numImage, CPU_UCHAR* pImage)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    int numElementImage = (iDecWidth / width) * (iDecHeight / height);
    CUDA_UCHAR** ppDecOutCUDA = new CUDA_UCHAR * [numImage];
    CUDA_UCHAR* pImageCUDA;
    for (int i = 0; i < numImage; i++) {
        hipMalloc((void**)&ppDecOutCUDA[i], iDecWidth * iDecHeight * 3 * sizeof(CUDA_UCHAR));
        hipMemcpy(ppDecOutCUDA[i], ppDecOut[i], iDecWidth * iDecHeight * 3 * sizeof(CUDA_UCHAR), hipMemcpyHostToDevice);
    }
    hipMalloc((void**)&pImageCUDA, width * height * 3 * numCam * sizeof(CUDA_UCHAR));

    int* pLookupImage = new int[numImage];
    MakeLookupTableImage(iDecWidth, iDecHeight, numCam, width, height, pLookupImage);

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(width, BLOCKDIM_X), iDivUp(height, BLOCKDIM_Y));

    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Invoke\n");

    for (int i = 0; i < numImage; i++) {
        convertDecodedToImageBuffer << <grid, threads >> > (ppDecOutCUDA[i], pImageCUDA, width, height, iDecWidth, iDecHeight, numCam, pLookupImage[i]);
    }
    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Finish\n");

    hipMemcpy(pImage, pImageCUDA, numCam * width * height * 3 * sizeof(CUDA_UCHAR), hipMemcpyDeviceToHost);
    for (int i = 0; i < numImage; i++) {
        hipFree(ppDecOutCUDA[i]);
    }
    hipFree(pImageCUDA);

    delete[]ppDecOutCUDA;
    delete[]pLookupImage;
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");


}
void SJPlenopticPacker::UnPackingImageWithIndex(SJDim numCam, SJDim width, SJDim height, const CPU_UCHAR** ppDecOut, int iDecWidth, int iDecHeight, int numImage, int index, CPU_UCHAR* pImage)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    int numElementImage = (iDecWidth / width) * (iDecHeight / height);
    int numImageID = index / numElementImage;
    int numElementID = index % numElementImage;
    CUDA_UCHAR* pDecOutCUDA;
    CUDA_UCHAR* pImageCUDA;
    hipMalloc((void**)&pDecOutCUDA, iDecWidth * iDecHeight * 3 * sizeof(CUDA_UCHAR));
    hipMalloc((void**)&pImageCUDA, width * height * 3 * numElementImage * sizeof(CUDA_UCHAR));

    hipMemcpy(pDecOutCUDA, ppDecOut[numImageID], iDecWidth * iDecHeight * 3 * sizeof(CUDA_UCHAR), hipMemcpyHostToDevice);

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(width, BLOCKDIM_X), iDivUp(height, BLOCKDIM_Y));

    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Invoke\n");

    convertDecodedToImageBuffer << <grid, threads >> > (pDecOutCUDA, pImageCUDA, width, height, iDecWidth, iDecHeight, numCam, 0);

    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Finish\n");


    hipMemcpy(pImage, &pImageCUDA[numElementID * width * height * 3], width * height * 3 * sizeof(CUDA_UCHAR), hipMemcpyDeviceToHost);

    hipFree(pDecOutCUDA);
    hipFree(pImageCUDA);
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}
void SJPlenopticPacker::UnPackingImageWithIndexCUDA(SJDim numCam, SJDim width, SJDim height, const CUDA_UCHAR** ppDecOut, int iDecWidth, int iDecHeight, int numImage, int index, CPU_UCHAR* pImage)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    int numElementImage = (iDecWidth / width) * (iDecHeight / height);
    int numImageID = index / numElementImage;
    int numElementID = index % numElementImage;
    CUDA_UCHAR* pImageCUDA;

    hipMalloc((void**)&pImageCUDA, width * height * 3 * numElementImage * sizeof(CUDA_UCHAR));

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(width, BLOCKDIM_X), iDivUp(height, BLOCKDIM_Y));

    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Invoke\n");

    convertDecodedToImageBuffer << <grid, threads >> > (ppDecOut[numImageID], pImageCUDA, width, height, iDecWidth, iDecHeight, numCam, 0);

    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Finish\n");


    hipMemcpy(pImage, &pImageCUDA[numElementID * width * height * 3], width * height * 3 * sizeof(CUDA_UCHAR), hipMemcpyDeviceToHost);

    hipFree(pImageCUDA);
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}
void SJPlenopticPacker::UnPackingLayerCPU(SJDim numCam, SJDim layerWidth, SJDim layerHeight, SJDim layerLevel, const CPU_UCHAR** ppDecOut, int iDecWidth, int iDecHeight, int numLayer, CPU_UCHAR* pLayer)
{
    LOGGING(LOG_LEVEL::VERBOSE, "Start\n");

    CUDA_UCHAR** ppDecOutCUDA = new CUDA_UCHAR * [numLayer];
    CUDA_UCHAR* pLayerCUDA;

    for (int i = 0; i < numLayer; i++) {
        hipMalloc((void**)&ppDecOutCUDA[i], iDecWidth * iDecHeight * 3 * sizeof(CUDA_UCHAR));
        hipMemcpy(ppDecOutCUDA[i], ppDecOut[i], iDecWidth * iDecHeight * 3 * sizeof(CUDA_UCHAR), hipMemcpyHostToDevice);
    }
    hipMalloc((void**)&pLayerCUDA, layerWidth * layerHeight * layerLevel * numCam * sizeof(CUDA_UCHAR));

    int* pLookupAlphaImage = new int[numLayer];
    int* pLookupAlphaLevel = new int[numLayer];
    MakeLookupTableLayer(iDecWidth, iDecHeight, numCam, layerWidth, layerHeight, layerLevel, pLookupAlphaImage, pLookupAlphaLevel);

    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(layerWidth, BLOCKDIM_X), iDivUp(layerHeight, BLOCKDIM_Y));

    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Invoke\n");

    for (int i = 0; i < numLayer; i++) {
        convertDecodedToAlphaBuffer << <grid, threads >> > (ppDecOutCUDA[i], pLayerCUDA, layerWidth, layerHeight, iDecWidth, iDecHeight, layerLevel, numCam, pLookupAlphaImage[i], pLookupAlphaLevel[i]);
    }

    LOGGING(LOG_LEVEL::VERBOSE, "CUDA Kernel Finish\n");

    hipMemcpy(pLayer, pLayerCUDA, numCam * layerWidth * layerHeight * layerLevel * sizeof(CUDA_UCHAR), hipMemcpyDeviceToHost);

    for (int i = 0; i < numLayer; i++) {
        hipFree(ppDecOutCUDA[i]);
    }
    hipFree(pLayerCUDA);

    delete[]ppDecOutCUDA;
    delete[]pLookupAlphaImage;
    delete[]pLookupAlphaLevel;
    LOGGING(LOG_LEVEL::VERBOSE, "End\n");

}

